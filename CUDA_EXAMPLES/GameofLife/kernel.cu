#include "hip/hip_runtime.h"
#include "common.h"
#include "cpu_anim.h"

#define DIM 500
#define INDEX(x, y) ((x)+(y)*DIM)

// Warto�ci wymagane przez procedur� aktualizuj�c�
struct DataBlock {
    unsigned char   *dev_bitmap;
	unsigned char   *dev_pom_bitmap;
	CPUAnimBitmap  *bitmap;
};

__device__ int step( int i, int j, unsigned char *col ) {

	int aliveNeighbours = 0;
	if (i != 0 && j != 0)
		aliveNeighbours += (col[4*INDEX(i - 1,j - 1)]) ? 1 : 0;
	if (i != 0)
	{
		aliveNeighbours += (col[4*INDEX(i - 1,j)]) ? 1 : 0;
		if (j+1 < DIM)
		aliveNeighbours += (col[4*INDEX(i - 1,j + 1)]) ? 1 : 0;
	}
	if (j != 0)
	{
		if (i+1 < DIM)	
		aliveNeighbours += (col[4*INDEX(i + 1,j - 1)]) ? 1 : 0;
		aliveNeighbours += (col[4*INDEX(i,j - 1)]) ? 1 : 0;
	}
	if (i+1 < DIM && j+1<DIM)
		aliveNeighbours += (col[4*INDEX(i + 1,j + 1)]) ? 1 : 0;
	if (j+1 < DIM)
		aliveNeighbours += (col[4*INDEX(i,j + 1)]) ? 1 : 0;
	if (i+1 < DIM)	
		aliveNeighbours += (col[4*INDEX(i + 1,j)]) ? 1 : 0;

	if (col[4*INDEX(i,j)] && aliveNeighbours > 1 && aliveNeighbours < 4)
		return aliveNeighbours;
	if (!col[4*INDEX(i,j)] && aliveNeighbours > 2 && aliveNeighbours < 4)
		return aliveNeighbours;
	return 0;
 
}

__global__ void kernel( unsigned char *ptr, unsigned char *pom, int t ) {
    // Odwzorowanie z blockIdx na po�o�enie piksela
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // Obliczenie warto�ci dla tego miejsca
    int isAlive = step( x, y, ptr );
    pom[offset*4 + 0] = 255 * isAlive;	//Red
    pom[offset*4 + 1] = 80 * isAlive;	//Green
    pom[offset*4 + 2] = t%80  * isAlive;	//Blue
    pom[offset*4 + 3] = 255 * isAlive;	//Alpha
}

//ustawia szachownice
__global__ void setBoard( unsigned char *ptr ) {
    // Odwzorowanie z blockIdx na po�o�enie piksela
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // Obliczenie warto�ci dla tego miejsca
    int isAlive = (offset % 2 + y % 2) %2;
    ptr[offset*4 + 0] = 255 * isAlive;	//Red
    ptr[offset*4 + 1] = 255 * isAlive;	//Green
    ptr[offset*4 + 2] = 255 * isAlive;	//Blue
    ptr[offset*4 + 3] = 255 * isAlive;
}

void generate_frame( DataBlock *d, int ticks )
{
	 hipEvent_t     start, stop;
	HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
	HANDLE_ERROR( hipEventRecord( start, 0 ) );

	dim3    grid(DIM,DIM);
	if (ticks == 1)
		setBoard<<<grid,1>>>( d->dev_bitmap );

    kernel<<<grid,1>>>( d->dev_bitmap, d->dev_pom_bitmap, ticks );

    HANDLE_ERROR( hipMemcpy( d->dev_bitmap, d->dev_pom_bitmap,
                              d->bitmap->image_size(),
                              hipMemcpyDeviceToDevice ) );

    HANDLE_ERROR( hipMemcpy( d->bitmap->get_ptr(), d->dev_bitmap,
                              d->bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );          

	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
	printf( "Czas generowania klatki:\t  %3.1f ms\n",
            elapsedTime  );

	HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
}
// Zwolnienie pami�ci na GPU
void cleanup( DataBlock *d ) {
    HANDLE_ERROR( hipFree( d->dev_bitmap ) ); 
	HANDLE_ERROR( hipFree( d->dev_pom_bitmap ) ); 
}



int main( void ) {
    DataBlock   data;
//    CPUBitmap bitmap( DIM, DIM, &data );
	CPUAnimBitmap  bitmap( DIM, DIM, &data );
    unsigned char    *dev_bitmap;
	unsigned char    *dev_pom_bitmap;
	data.bitmap = &bitmap;
    HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap, bitmap.image_size() ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_pom_bitmap, bitmap.image_size() ) );
    data.dev_bitmap = dev_bitmap;
	data.dev_pom_bitmap = dev_pom_bitmap;
       
	
    bitmap.anim_and_exit( (void (*)(void*,int))generate_frame,
                          (void (*)(void*))cleanup );
}

