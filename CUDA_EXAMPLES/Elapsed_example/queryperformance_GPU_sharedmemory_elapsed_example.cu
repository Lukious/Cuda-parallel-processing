#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <io.h>
#include <fcntl.h>
#include <sys/stat.h>
#include <Windows.h>
#include <time.h>
#include <chrono>

#define GRIDSIZE 8*1024
#define BLOCKSIZE 1024
#define TOTALSIZE (GRIDSIZE*BLOCKSIZE)

void genData(float* ptr, unsigned int size) {
    while (size--) {
        *ptr++ = (float)(rand() % 1000) / 1000.0F; 
    }
}

__global__ void adjDiff(float* result, float* input) {
    __shared__ float s_data[BLOCKSIZE];
    unsigned int tx = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    s_data[tx] = input[i];

    __syncthreads();

    if (tx > 0) {
        result[i] = s_data[tx] - s_data[tx - 1];
        //Calculate and store the result to global memory
    }
    else if (i > 0) {
        result[i] = s_data[tx] - input[i - 1];
    }
}

__host__ int main(void) {

    float* pSource = NULL;
    float* pResult = NULL;
    int i;

    long long cntStart = 0LL, cntEnd = 0LL, freq = 0LL;


    QueryPerformanceFrequency((LARGE_INTEGER*)(&freq));


    pSource = (float*)malloc(TOTALSIZE * sizeof(float));
    pResult = (float*)malloc(TOTALSIZE * sizeof(float));

    genData(pSource, TOTALSIZE);

    float* pSourceDev = NULL;
    float* pResultDev = NULL;

    pResult[0] = 0.0F; //exceptional case for i = 0
    hipMalloc((void**)&pSourceDev, TOTALSIZE * sizeof(float));
    hipMalloc((void**)&pResultDev, TOTALSIZE * sizeof(float));

    //QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart));

    hipMemcpy(pSourceDev, pSource, TOTALSIZE * sizeof(float), hipMemcpyHostToDevice);

    //start the timer
    QueryPerformanceCounter((LARGE_INTEGER*)(&cntStart));
    //calculate the adjacent differnce
    // adjDiff(pResult, pSource, TOTALSIZE);

    ///------------------------------------------------------------------
    dim3 dimGrid(GRIDSIZE, 1, 1);
    dim3 dimBlock(BLOCKSIZE, 1, 1);
    adjDiff << <dimGrid, dimBlock >> > (pResultDev, pSourceDev);

    ///------------------------------------------------------------------

    //end the timer 
    QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd));

    hipMemcpy(pResult, pResultDev, TOTALSIZE * sizeof(float), hipMemcpyDeviceToHost);

    // QueryPerformanceCounter((LARGE_INTEGER*)(&cntEnd));

    printf("elapsed time  = %f usec\n", (double)(cntEnd - cntStart) * 1000000.0 / (double)(freq));

    i = 1;
    printf("i = %7d : %f = %f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    i = TOTALSIZE - 1;
    printf("i = %7d : %f = %f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    i = TOTALSIZE / 2;
    printf("i = %7d : %f = %f-%f\n", i, pResult[i], pSource[i] , pSource[i - 1]);


    free(pSource);
    free(pResult);
    hipFree(pSourceDev);
    hipFree(pResultDev);
}

